#include "hip/hip_runtime.h"
#include <unistd.h>
#include <iostream>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "fp16_conversion.h"

using namespace std;

//------------------------------------------------------------run time----------------------------------------//
//如果需要运行10个小时，只需要把#define hours (xxx)改成#define hours (10)就可以了
#define hours (0.04)

#include <time.h>
typedef long time_t;
#define seconds (3600)
#define timeover (hours*seconds)
//------------------------------------------------------------run time----------------------------------------//

//-----------------------------------------------------------------csv----------------------------------------//
typedef struct record
{
    char id1[30];
    char id2[30];
    char id3[30];
    char id4[30];
    char id5[30];
    char id6[30];
}rec;
//---------------------------------------------------------------csv----------------------------------------//


const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
    }
    return "unknown error";
}

inline
hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

inline
hipblasStatus_t checkCublas(hipblasStatus_t result)
{
  if (result != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cublasGetErrorString(result));
    assert(result == HIPBLAS_STATUS_SUCCESS);
  }
  return result;
}

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on CPU
void CPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	int a=1;

    for(int i = 0; i < nr_rows_A * nr_cols_A; i++){
		A[i] = (float)rand()/(float)(RAND_MAX/a);
	}
}

int main(int argc, char ** argv){
     long count = 0;
     long run_time;
     long minutes=0;
     long times=1;//矩阵的倍数，真实大小为1024乘以times，对不同的设备，times的数值不同
     time_t time_begin,time_end;
     sscanf(argv[1], "%ld", &run_time);
     sscanf(argv[2], "%ld", &times);
     printf("----------------------------start-------------------------------\n");

    //-----------------------------------------------------------csv----------------------------------------//
    //统计，性能，耗时，数组大小，矩阵大小，运算次数，平均速度，十亿次/秒，每
    rec item[3]={
        //{"statistics,","performance,","Time,","size,","matrix size,","average,"}
        {"统计,","性能(GFlop/s),","耗时(s),","运算次数,","数组大小,","平均耗时,"}
    };

    //rec r[3]={
    //    {"0001,","zhaoge,","30,","98,","98,","98,"},
    //    {"0002,","fenghao,","24,","60,","98,","98,"}
    //};

    rec *p;   
    FILE *T0;

    T0=fopen("benchmark_record/benchmark_record2.csv","a");
    p = item;
    fwrite(p->id1,1,strlen(p->id1),T0);
    fwrite(p->id2,1,strlen(p->id2),T0);
    fwrite(p->id3,1,strlen(p->id3),T0);
    //fwrite(p->id4,1,strlen(p->id4),T0);
    //fwrite(p->id5,1,strlen(p->id5),T0);
    //fwrite(p->id6,1,strlen(p->id6),T0);
     fwrite("\r\n",1,3,T0);
    char printfloat16[10] = "float16,";
    char printfloat32[10] = "float32,";
    char performance[8];
    char costTime[7];
    //char average[9];

    //-----------------------------------------------------------csv----------------------------------------//


  //int min_m_k_n = 2;
  //int max_m_k_n = 4096*8;
  int max_m_k_n = 1024 * times;
  //int max_m_k_n = 1024 * 6;//nano上6或8都合适
  int repeats = 10;
  int verbose = 1;

  
  if(verbose) 
    cout << "运行的数组大小为： "  << max_m_k_n
	 //<< " 每次结果的运行次数: " << repeats
	 << endl;

  hipblasStatus_t stat;
  hipblasHandle_t handle;

  checkCublas(hipblasCreate(&handle));

  //if(verbose) cout << "allocating device variables" << endl;
  if(verbose) cout << "正在分配设备变量" << endl;
  
  // Allocate 3 arrays on CPU
  
  float *h_A = (float *)malloc(max_m_k_n * max_m_k_n * sizeof(float));
  float *h_B = (float *)malloc(max_m_k_n * max_m_k_n * sizeof(float));
  float *h_C = (float *)malloc(max_m_k_n * max_m_k_n * sizeof(float));
  
  CPU_fill_rand(h_A, max_m_k_n, max_m_k_n);
  CPU_fill_rand(h_B, max_m_k_n, max_m_k_n);
  CPU_fill_rand(h_C, max_m_k_n, max_m_k_n);

    // Allocate 3 arrays on GPU
    float *d_A32, *d_B32, *d_C32;
    checkCuda(hipMallocManaged(&d_A32, max_m_k_n * max_m_k_n * sizeof(float)));
    checkCuda(hipMallocManaged(&d_B32, max_m_k_n * max_m_k_n * sizeof(float)));
    checkCuda(hipMallocManaged(&d_C32, max_m_k_n * max_m_k_n * sizeof(float)));
    
    checkCuda(hipMemcpy(d_A32,h_A,max_m_k_n * max_m_k_n * sizeof(float),hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_B32,h_B,max_m_k_n * max_m_k_n * sizeof(float),hipMemcpyHostToDevice));
    checkCuda(hipMemcpy(d_C32,h_C,max_m_k_n * max_m_k_n * sizeof(float),hipMemcpyHostToDevice));
    
    int lda, ldb, ldc, m, n, k;
    const float alf32 = 1.0f;
    const float bet32 = 0.0f;
    const float *alpha32 = &alf32;
    const float *beta32 = &bet32;
  
    
  	__half *d_A16, *d_B16, *d_C16;
    checkCuda(hipMallocManaged(&d_A16, max_m_k_n * max_m_k_n * sizeof(__half)));
    checkCuda(hipMallocManaged(&d_B16, max_m_k_n * max_m_k_n * sizeof(__half)));
    checkCuda(hipMallocManaged(&d_C16, max_m_k_n * max_m_k_n * sizeof(__half)));
    
    for (int i = 0; i < max_m_k_n * max_m_k_n; i++) {
      d_A16[i] = approx_float_to_half(h_A[i]);
  	  d_B16[i] = approx_float_to_half(h_B[i]);
  	  d_C16[i] = approx_float_to_half(h_C[i]);
    }
    
    //int lda, ldb, ldc, m, n, k;
    const __half alf16 = approx_float_to_half(1.0);
    const __half bet16 = approx_float_to_half(0.0);
    const __half *alpha16 = &alf16;
    const __half *beta16 = &bet16;

  //-------------------------------------------------------------------------------------------------//  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  int size = max_m_k_n;
  bool is32 = true;
 
  time(&time_begin);
  while(1){
	  if(is32 == true)  
          {
             printf("------------------------------float16---------------------------\n");
             is32 = false;
           }
	  else  
          {
            printf("-------------------------------float32--------------------------\n");
            is32 = true;
          }
	  //if(count == 4)
	  //	  break;
          time(&time_end);
	  count = time_end - time_begin;
          if(count > run_time)
                //break;
             { 
                //printf("count: %d\n",count);
                minutes = count/60;
                printf("设置的总运行时间(分钟): %ld\n",minutes);
                //printf("time time_end: %d\n",time_end);
                break;
            }
	  //count += 1;
    	  double sum = 0.0;
          double avg = 0.0;
    	  for(int rep = 0; rep < repeats; rep++){
      		hipEventRecord(start, 0);
	  	m=n=k=size;
	  	lda = m;
	  	ldb = k;
	  	ldc = m;
	  	if(is32 == true)
        		stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha32, d_A32, lda, d_B32, ldb, beta32, d_C32, ldc); 
	  	else
			stat = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha16, d_A16, lda, d_B16, ldb, beta16, d_C16, ldc); 
      		hipEventRecord(stop,0);
      		hipEventSynchronize(stop);
      		if(stat != HIPBLAS_STATUS_SUCCESS){
			cerr << "hipblasSgemmBatched failed" << endl;
			exit(1);
      		}
      		assert(!hipGetLastError());
      
      		float elapsed;
      		hipEventElapsedTime(&elapsed, start, stop);
      		elapsed /= 1000.0f;
      		sum += elapsed;
    		}
	
        avg = 	sum/repeats;
	 if(is32 == true)
  		//cout << "float32; size " << size << " average: " << avg << " s "<< endl;
                {
                cout << "float32; 数组大小 " << size << " 平均速度: " << avg << " s "<< endl;
                minutes = run_time/60;
                printf("设置的总运行时间(分钟): %ld\n",minutes);
                minutes = count/60;
                printf("目前已运行时间(分钟): %ld\n",minutes);
                //printf("运行时间(秒): %ld\n",run_time);
		//printf("从开始到目前运行总时间(秒): %ld\n",count);
                 }
	 else
  		//cout << "float16; size " << size << " average: " << sum/repeats << " s "<< endl;
                {
                cout << "float16; 数组大小 " << size << " 平均速度: " << sum/repeats << " s "<< endl;
                //printf("运行时间(秒): %ld\n",run_time);
                 }
         //printf("----------------------------------------------------------------\n");

  //-----------------------added by shihailong-----------------------------//
   	float msecPerMatrixMul = sum/repeats;
   	double flopsPerMatrixMul = 2.0 * (double)size * (double)size * (double)size;
   //double flopsPerMatrixMul = 2.0 * (double)max_m_k_n * (double)max_m_k_n * (double)max_m_k_n;
   	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul);
        printf(
            //"Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops\n\n",
            "性能= %.2f GFlop/s, 耗时= %.3f s, 运算次数= %.0f Ops\n\n",
            gigaFlops,
            msecPerMatrixMul,
            flopsPerMatrixMul);

//-----------------------------csv--------------------------------------------------------------//
        //p = r;
        if(is32 == true)
             fwrite(printfloat32,1,10,T0);
        else
             fwrite(printfloat16,1,10,T0);

        //fwrite(&gigaFlops,8,1,T0);
        sprintf(performance,"%.2f,", gigaFlops);
        //fwrite(&performance,1,10,T0);
        fwrite(&performance,1,sizeof(performance),T0);
        //fwrite(p->id2,1,strlen(p->id2),T0);

         sprintf(costTime,"%.3f,", msecPerMatrixMul);
        fwrite(&costTime,1,sizeof(costTime),T0);
        //fwrite(p->id3,1,strlen(p->id3),T0);



        //fwrite(p->id4,1,strlen(p->id4),T0);
        //fwrite(p->id5,1,strlen(p->id5),T0);

        //sprintf(average,"%.5f,", avg);
        //fwrite(&average,1,sizeof(average),T0);
        //fwrite(p->id6,1,strlen(p->id6),T0);
        fwrite("\r\n",1,3,T0);
    //fclose(T0);
        
//-----------------------------csv--------------------------------------------------------------//
  }

fclose(T0);

  	hipFree(d_A32);
  	hipFree(d_B32);
  	hipFree(d_C32);
  	hipFree(d_A16);
  	hipFree(d_B16);
  	hipFree(d_C16);

  // Free CPU memory
  free(h_A);
  free(h_B);
  free(h_C);
      
  return 0;
}
